#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <cuda/pipeline>

#include "async_copies.h"
#include "wmma_extension.h"

using namespace nvcuda;

#ifndef COMPLEX
#define COMPLEX 2
#endif
#ifndef REAL
#define REAL 0
#endif
#ifndef IMAG
#define IMAG 1
#endif

// All values related to data layout must be defined at compile time
#if !defined M || !defined _N || !defined K
#error                                                                         \
    "M, _N, K and values per block, warp, tensor core must be defined at compile time"
#endif

#if NBIT == 16
using Tin = half;
using Ttc = half;
using Tout = float;
#else
#error NBIT must be 16
#endif

// basic data layout
using A_t = Tin[COMPLEX][M][K];
using B_t = Tin[COMPLEX][_N][K];
// data layout for optimal transfer to shared memory
using A_opt_t =
    Tin[M / M_PER_BLOCK][K / K_PER_WMMA][COMPLEX][M_PER_BLOCK][K_PER_WMMA];
using B_opt_t =
    Tin[_N / N_PER_BLOCK][K / K_PER_WMMA][COMPLEX][N_PER_BLOCK][K_PER_WMMA];
using C_t = Tout[COMPLEX][M][_N];

extern "C" __global__ void wmma_complex_gemm_basic(C_t C, const A_t A,
                                                   const B_t B) {
  const unsigned blockN = blockIdx.x;
  const unsigned blockM = blockIdx.y;
  const unsigned warpN = threadIdx.y;
  const unsigned warpM = threadIdx.z;

  // number of tiles processed by each warp
  constexpr unsigned M_TILES = M_PER_WARP / M_PER_WMMA;
  constexpr unsigned N_TILES = N_PER_WARP / N_PER_WMMA;
  constexpr unsigned K_TILES = K / K_PER_WMMA;

  wmma::fragment<wmma::accumulator, M_PER_WMMA, N_PER_WMMA, K_PER_WMMA, Tout>
      sum[COMPLEX][M_TILES][N_TILES];
  for (int c = 0; c < COMPLEX; c++) {
    for (int m = 0; m < M_TILES; m++) {
      for (int n = 0; n < N_TILES; n++) {
        wmma::fill_fragment(sum[c][m][n], 0);
      }
    }
  }

  for (int k = 0; k < K_TILES; k++) {
    // declare input fragments
    wmma::fragment<wmma::matrix_a, M_PER_WMMA, N_PER_WMMA, K_PER_WMMA, Ttc,
                   wmma::row_major>
        a[COMPLEX][M_TILES];
    wmma::fragment<wmma::matrix_b, M_PER_WMMA, N_PER_WMMA, K_PER_WMMA, Ttc,
                   wmma::col_major>
        b[COMPLEX][N_TILES];

    // load matrices from global memory
    for (int c = 0; c < COMPLEX; c++) {
      for (int m = 0; m < M_TILES; m++) {
        int k_index = k * K_PER_WMMA;
        wmma::load_matrix_sync(a[c][m],
                               &A[c][blockM * M_PER_BLOCK + warpM * M_PER_WARP +
                                     m * M_PER_WMMA][k_index],
                               K);
      }
    }

    for (int c = 0; c < COMPLEX; c++) {
      for (int n = 0; n < N_TILES; n++) {
        int k_index = k * K_PER_WMMA;
        wmma::load_matrix_sync(b[c][n],
                               &B[c][blockN * N_PER_BLOCK + warpN * N_PER_WARP +
                                     n * N_PER_WMMA][k_index],
                               K);
      }
    }

    // step 1 and 2
    for (int m = 0; m < M_TILES; m++) {
      for (int n = 0; n < N_TILES; n++) {
        wmma::mma_sync(sum[REAL][m][n], a[REAL][m], b[REAL][n],
                       sum[REAL][m][n]);
        wmma::mma_sync(sum[IMAG][m][n], a[REAL][m], b[IMAG][n],
                       sum[IMAG][m][n]);
      }
    }

    // step 3
    __syncwarp();
    for (int n = 0; n < N_TILES; n++) {
      for (auto &element : b[IMAG][n].x) {
        element = -element;
      }
    }
    __syncwarp();

    // step 4 and 5
    for (int m = 0; m < M_TILES; m++) {
      for (int n = 0; n < N_TILES; n++) {
        wmma::mma_sync(sum[REAL][m][n], a[IMAG][m], b[IMAG][n],
                       sum[REAL][m][n]);
        wmma::mma_sync(sum[IMAG][m][n], a[IMAG][m], b[REAL][n],
                       sum[IMAG][m][n]);
      }
    }
  }

  // store the result to global memory
  for (int c = 0; c < COMPLEX; c++) {
    for (int m = 0; m < M_TILES; m++) {
      for (int n = 0; n < N_TILES; n++) {
        Tout *c_ptr =
            &C[c][blockM * M_PER_BLOCK + warpM * M_PER_WARP + m * M_PER_WMMA]
              [blockN * N_PER_BLOCK + warpN * N_PER_WARP + n * N_PER_WMMA];
        wmma::store_matrix_sync(c_ptr, sum[c][m][n], _N, wmma::mem_row_major);
      }
    }
  }
}

extern "C" __global__ void wmma_complex_gemm_opt(C_t C, const A_opt_t A,
                                                 const B_opt_t B) {
  const unsigned blockN = blockIdx.x;
  const unsigned blockM = blockIdx.y;
  const unsigned warpN = threadIdx.y;
  const unsigned warpM = threadIdx.z;

  constexpr unsigned num_threads = block_size_x * block_size_y * block_size_z;
  const unsigned tid = threadIdx.x + threadIdx.y * block_size_x +
                       threadIdx.z * block_size_x * block_size_y;

  // number of tiles processed by each warp
  constexpr unsigned M_TILES = M_PER_WARP / M_PER_WMMA;
  constexpr unsigned N_TILES = N_PER_WARP / N_PER_WMMA;
  constexpr unsigned K_TILES = K / K_PER_WMMA;

  // initialize accumulator fragments to zero
  wmma::fragment<wmma::accumulator, M_PER_WMMA, N_PER_WMMA, K_PER_WMMA, Tout>
      sum[COMPLEX][M_TILES][N_TILES];
  for (int c = 0; c < COMPLEX; c++) {
    for (int m = 0; m < M_TILES; m++) {
      for (int n = 0; n < N_TILES; n++) {
        wmma::fill_fragment(sum[c][m][n], 0);
      }
    }
  }

  // shared memory buffers for partial A and B matrix. Several buffers to allow
  // for async operations: copy next submatrix to shared memory while computing
  // current submatrix
  __shared__ Tin A_s[NBUFFER][COMPLEX][M_PER_BLOCK][K_PER_WMMA];
  __shared__ Tin B_s[NBUFFER][COMPLEX][N_PER_BLOCK][K_PER_WMMA];

  cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();

  for (unsigned k = 0, f = 0; k < K_TILES; ++k) {

    // declare input fragments for A and B matrices
    wmma::fragment<wmma::matrix_a, M_PER_WMMA, N_PER_WMMA, K_PER_WMMA, Ttc,
                   wmma::row_major>
        a[COMPLEX][M_TILES];
    wmma::fragment<wmma::matrix_b, M_PER_WMMA, N_PER_WMMA, K_PER_WMMA, Ttc,
                   wmma::col_major>
        b[COMPLEX][N_TILES];

    // copy next data to smem
    for (; f < K_TILES && f < (k + NBUFFER); ++f) {
      pipe.producer_acquire();
      // trick: next buffer to load is always the one previous to current loop
      // the % operation only works if k is unsigned
      copy_async<sizeof(A_s[0]), num_threads>(&A_s[f % NBUFFER][0][0],
                                              &A[blockM][f][0][0],
                                              pipe, tid);
      copy_async<sizeof(B_s[0]), num_threads>(&B_s[f % NBUFFER][0][0],
                                              &B[blockN][f][0][0],
                                              pipe, tid);
      pipe.producer_commit();
    }

    // NBUFFER copy operations have been started
    // the oldest one needs to be finished before we can start computation on
    // that data This corresponds to (NBUFFER - 1) copy operations ago so that
    // is the one we need to wait for
    cuda::pipeline_consumer_wait_prior<NBUFFER - 1>(pipe);
    __syncthreads(); // not sure if this is needed, perhaps already handled by
                     // the pipe.wait_prior

    // load A matrix from shared memory
    for (int c = 0; c < COMPLEX; c++) {
      for (int m = 0; m < M_TILES; m++) {
        wmma::load_matrix_sync(
            a[c][m],
            &A_s[k % NBUFFER][c][warpM * M_PER_WARP + m * M_PER_WMMA][0],
            K_PER_WMMA);
      }
    }

    // load B matrix from shared memory
    for (int c = 0; c < COMPLEX; c++) {
      for (int n = 0; n < N_TILES; n++) {
        wmma::load_matrix_sync(
            b[c][n],
            &B_s[k % NBUFFER][c][warpN * N_PER_WARP + n * N_PER_WMMA][0],
            K_PER_WMMA);
      }
    }

    // do the MMA
    // In general, MMA is D = A x B + C
    // Here, D and C are the same matrix so we have C += A x B
    // steps to do complex MMA with separate real and imaginary data
    // with x == x_r + x_i * i
    // a * b = (a_r * b_r - a_i * b_i) + (a_r * b_i + a_i * b_r) * i
    // 1. sum[real] += a_r * b_r
    // 2. sum[imag] += a_r * b_i
    // 3. b_i = - b_i (because tensor cores cannot subtract)
    // 4. sum[real] += a_i * b_i
    // 5. sum[imag] += a_i * b_r

    // steps 1 and 2
    for (int m = 0; m < M_TILES; m++) {
      for (int n = 0; n < N_TILES; n++) {
        wmma::mma_sync(sum[REAL][m][n], a[REAL][m], b[REAL][n],
                       sum[REAL][m][n]);
        wmma::mma_sync(sum[IMAG][m][n], a[REAL][m], b[IMAG][n],
                       sum[IMAG][m][n]);
      }
    }

    // step 3
    __syncwarp();
    for (int n = 0; n < N_TILES; n++) {
      for (auto &element : b[IMAG][n].x) {
        element = -element;
      }
    }
    __syncwarp();

    // step 4 and 5
    for (int m = 0; m < M_TILES; m++) {
      for (int n = 0; n < N_TILES; n++) {
        wmma::mma_sync(sum[REAL][m][n], a[IMAG][m], b[IMAG][n],
                       sum[REAL][m][n]);
        wmma::mma_sync(sum[IMAG][m][n], a[IMAG][m], b[REAL][n],
                       sum[IMAG][m][n]);
      }
    }

    pipe.consumer_release();

    __syncthreads();
  }

  // store the result to global memory
  for (int c = 0; c < COMPLEX; c++) {
    for (int m = 0; m < M_TILES; m++) {
      for (int n = 0; n < N_TILES; n++) {
        Tout *c_ptr =
            &C[c][blockM * M_PER_BLOCK + warpM * M_PER_WARP + m * M_PER_WMMA]
              [blockN * N_PER_BLOCK + warpN * N_PER_WARP + n * N_PER_WMMA];
        wmma::store_matrix_sync(c_ptr, sum[c][m][n], _N, wmma::mem_row_major);
      }
    }
  }
}
